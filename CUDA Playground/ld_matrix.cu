#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_fp8.h>
#include <vector>
#include <ctime>
#include <random>

template<typename T, int M, int N>
struct ptx_fragment {
    T data[M * N / 32]; 
    uint32_t raw[M * N / 32];
    static constexpr int total_elements = M * N;
    static constexpr int total_bytes = M * N * sizeof(T);
    static constexpr int bytes_per_thread = total_bytes / 32;
    static constexpr int elements_per_thread = total_elements / 32;
};

__device__ __forceinline__ uint32_t load_matrix_ptx_8x8(__hip_fp8x4_e4m3_fnuz* shared_addr) {
    uint32_t result;
    uint32_t source = *reinterpret_cast<uint32_t*>(&shared_addr);
    asm volatile (
        "ldmatrix.sync.aligned.m8n8.x1.shared.b32 {%0}, [%1];"
        : "=r"(result)           // output: 32-bit register
        : "r"(shared_addr)       // input: 32-bit address register  
        );
    return result;
}

__global__ void kernel_launch(int M, int N, __hip_fp8x4_e4m3_fnuz* data) {
    ptx_fragment<__hip_fp8x4_e4m3_fnuz, 16, 16> frag;

    // Need 16x16 = 256 fp8 elements = 64 __hip_fp8x4_e4m3_fnuz elements
    __shared__ __hip_fp8x4_e4m3_fnuz shared_data[16];

    // Load enough data (assuming you have it)
    for (int i = 0; i < 64 && i < M * N / 4; i++) {
        shared_data[i] = data[i];
    }


    uint32_t res = load_matrix_ptx_8x8(&shared_data[0]);

    }



int main(){
    std::mt19937            rng{ std::random_device{}() };
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    std::vector<__hip_fp8x4_e4m3_fnuz> mat;
    for (int i = 0; i < 16 * 16; i+=4) {
        float4 f4 = make_float4(1.0f, 1.0f, 1.0f, 1.0f);
        mat.push_back(__hip_fp8x4_e4m3_fnuz(f4));
    }
    __hip_fp8x4_e4m3_fnuz* d_data;
    
    hipMalloc(&d_data, 16 * 16);
    hipMemcpy(d_data, mat.data(), 16 * 16, hipMemcpyHostToDevice);

    kernel_launch << <1, 1 >> > (16, 16, d_data);

    hipFree(d_data);
    
}